#include "hip/hip_runtime.h"
#include "entropy.cuh"


#include <cmath>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include "cuda_utils/kerneldimscalculator.h"
#include "cuda_utils/threadindexing.cuh"
#include "cuda_utils/hip/hip_runtime_api.h"
#include "mean.cuh"

using namespace attribute;

using namespace std;
using namespace volume;



__global__ void _entropyKernel(const Volume* volume, float *data, int texSide) {
    __shared__ unsigned histogram[256];
    __shared__ float buffer[256];
    __shared__ int nVoxels;

    unsigned index = ThreadIndexing::globalIdx() / ThreadIndexing::totalBlockDim();

    if(index < volume->voxelCount()) {
        _calcHistogram(volume, (int)index, texSide, histogram, &nVoxels);

        _calcEntropy(histogram, nVoxels, buffer);

        if(ThreadIndexing::totalBlockIdx() == 0){
            *(data + index) = buffer[0];
        }
    }
}


__device__ void _calcEntropy(const unsigned *histogram, unsigned nVoxels, float *buffer) {
    unsigned tid = ThreadIndexing::totalBlockIdx();

    if(tid < ThreadIndexing::totalBlockDim()){
        float value = (float) histogram[tid] / (float) nVoxels;

        if(value != 0.0f){
            buffer[tid] = (float) value * log10(value);
        }
        else{
            buffer[tid] = 0;
        }
        __syncthreads();

        unsigned n = ThreadIndexing::totalBlockDim();
        unsigned halfN;

        while(n > 1) {
            halfN = n / 2;

            if (tid < halfN){
                buffer[tid] = buffer[tid] + buffer[tid + halfN];
            }
            __syncthreads();

            n = halfN;
        }
        if(tid == 0){
            buffer[tid] = - buffer[tid];
        }
    }

    __syncthreads();
}





Entropy::Entropy() : Attribute(){
    setType(AttributeType::ENTROPY);
    setDescription("Entropy");
}



Entropy::~Entropy() {

}


thrust::host_vector<float>* Entropy::kernelLaunch(const Volume* volume, size_t threadCount, dim3 blockDim, int cubeSide) {
    thrust::host_vector<float>* hostVector = new thrust::host_vector<float>(threadCount, 0);
    thrust::device_vector<float> devVector = *hostVector;

    blockDim = dim3(256, 1, 1);
    dim3 gridDim = KernelDimsCalculator::instance().calc(blockDim, threadCount * 256);

    _entropyKernel<<<gridDim, blockDim>>>(volume, thrust::raw_pointer_cast(&devVector[0]), cubeSide);
    getLastCudaError("Entropy kernel failed.");
    checkCudaErrors(hipDeviceSynchronize());

    *hostVector = devVector;

    return hostVector;
}
