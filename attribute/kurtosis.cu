#include "hip/hip_runtime.h"
#include "kurtosis.cuh"


#include <cmath>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include "cuda_utils/kerneldimscalculator.h"
#include "cuda_utils/threadindexing.cuh"
#include "cuda_utils/hip/hip_runtime_api.h"
#include "mean.cuh"
#include "standarddeviation.cuh"

using namespace attribute;

using namespace std;
using namespace volume;



__global__ void _kurtosisKernel(const Volume* volume, float *data, int texSide) {
    __shared__ unsigned histogram[256];
    __shared__ float buffer[256];
    __shared__ int nVoxels;

    unsigned index = ThreadIndexing::globalIdx() / ThreadIndexing::totalBlockDim();

    if(index < volume->voxelCount()) {
        _calcHistogram(volume, (int)index, texSide, histogram, &nVoxels);
        _calcMean(histogram, nVoxels, buffer);
        float mean = buffer[0];
        _calcStandardDeviation(histogram, nVoxels, buffer, mean);
        float standardDeviation = buffer[0];

        _calcKurtosis(histogram, nVoxels, buffer, mean, standardDeviation);

        if(ThreadIndexing::totalBlockIdx() == 0){
            *(data + index) = buffer[0];
        }
    }
}


__device__ void _calcKurtosis(const unsigned *histogram, unsigned nVoxels, float *buffer, float mean, float standadDeviation) {
    unsigned tid = ThreadIndexing::totalBlockIdx();


    if(tid <  ThreadIndexing::totalBlockIdx()){
        float value = tid  - mean;
        buffer[tid] = value * value * value * value * (float) histogram[tid];

        __syncthreads();

        unsigned n =  ThreadIndexing::totalBlockIdx();
        unsigned halfN;


        while(n > 1) {
            halfN = n / 2;

            if (tid < halfN){
                buffer[tid] = buffer[tid] + buffer[tid + halfN];
            }
            __syncthreads();

            n = halfN;
        }

        if(tid == 0){
            if(standadDeviation != 0) {
                buffer[tid] = buffer[tid] / nVoxels;
                buffer[tid] = buffer[tid] / (standadDeviation * standadDeviation * standadDeviation * standadDeviation) - 3;
            }
            else{
                buffer[tid] = 0;
            }
        }
    }

    __syncthreads();
}





Kurtosis::Kurtosis() : Attribute(){
    setType(AttributeType::KURTOSIS);
    setDescription("Kurtosis");
}



Kurtosis::~Kurtosis() {

}


thrust::host_vector<float>* Kurtosis::kernelLaunch(const Volume* volume, size_t threadCount, dim3 blockDim, int mask) {
    thrust::host_vector<float>* hostVector = new thrust::host_vector<float>(threadCount, 0);
    thrust::device_vector<float> devVector = *hostVector;

    blockDim = dim3(256, 1, 1);
    dim3 gridDim = KernelDimsCalculator::instance().calc(blockDim, threadCount * 256);

    _kurtosisKernel<<<gridDim, blockDim>>>(volume, thrust::raw_pointer_cast(&devVector[0]), mask);
    getLastCudaError("Kurtosis kernel failed.");
    checkCudaErrors(hipDeviceSynchronize());

    *hostVector = devVector;

    return hostVector;
}
