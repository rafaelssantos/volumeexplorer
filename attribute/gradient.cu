#include "hip/hip_runtime.h"
#include "gradient.cuh"

#include <cmath>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include "cuda_utils/kerneldimscalculator.h"
#include "cuda_utils/threadindexing.cuh"
#include "quaternion/quaternioncalculator.cuh"
#include "cuda_utils/hip/hip_runtime_api.h"

using namespace attribute;

using namespace quaternion;
using namespace std;
using namespace volume;




__global__ void _gradientKernel(const Volume* volume, float *data) {
    unsigned tid = ThreadIndexing::globalIdx();

    if(tid < volume->voxelCount()){
        *(data + tid) = _calcGradient(volume, (int) tid);
	}
}




__device__ float _calcGradient(const Volume* volume, int index){
	int3 voxelIndex = volume->indexToIndex3(index);

	float gX[3][3][3], gY[3][3][3], gZ[3][3][3];
	float hX[3] = {+1, +2, +1}, hY[3] = {+1, +2, +1}, hZ[3] = {+1, +2, +1};
	float hpX[3] = {+1, +0, -1}, hpY[3] = {+1, +0, -1}, hpZ[3] = {+1, +0,-1};

	for(int m = 0; m <= 2; m++){
		for(int n = 0; n <= 2; n++){
			for(int k = 0; k <= 2; k++){
				gY[m][n][k] = hpX[m] * hY[n] * hZ[k];
				gX[m][n][k] = hX[m] * hpY[n] * hZ[k];
				gZ[m][n][k] = hX[m] * hY[n] * hpZ[k];
			}
		}
	}


	float3 gradient = {0, 0, 0};

	for(int m = -1; m <= 1; m++){
		for(int n = -1; n <= 1; n++){
			for(int k = -1; k <= 1; k++){
				if(volume->isIndex3In(voxelIndex.x - m, voxelIndex.y - n, voxelIndex.z - k)){
                    gradient.x += gY[m+1][n+1][k+1] * static_cast<float>(volume->sampleAt(voxelIndex.x - m, voxelIndex.y - n, voxelIndex.z - k));
                    gradient.y += gX[m+1][n+1][k+1] * static_cast<float>(volume->sampleAt(voxelIndex.x - m, voxelIndex.y - n, voxelIndex.z - k));
                    gradient.z += gZ[m+1][n+1][k+1] * static_cast<float>(volume->sampleAt(voxelIndex.x - m, voxelIndex.y - n, voxelIndex.z - k));
				}
			}
		}
	}

    return QuaternionCalculator::norm(gradient);
}



Gradient::Gradient() : Attribute(){
    setType(AttributeType::GRADIENT);
    setDescription("Gradient");
}



Gradient::~Gradient() {

}


thrust::host_vector<float>* Gradient::kernelLaunch(const Volume* volume, size_t threadCount, dim3 blockDim, int) {
    thrust::host_vector<float>* hostVector = new thrust::host_vector<float>(threadCount, 0);
    thrust::device_vector<float> devVector = *hostVector;

    dim3 gridDim = KernelDimsCalculator::instance().calc(blockDim, threadCount);

	_gradientKernel<<<gridDim, blockDim>>>(volume, thrust::raw_pointer_cast(&devVector[0]));
    getLastCudaError("Gradient kernel failed.");
	checkCudaErrors(hipDeviceSynchronize());

	*hostVector = devVector;

	return hostVector;
}
