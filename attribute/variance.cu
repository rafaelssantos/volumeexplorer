#include "hip/hip_runtime.h"
#include "variance.cuh"


#include <cmath>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include "cuda_utils/kerneldimscalculator.h"
#include "cuda_utils/threadindexing.cuh"
#include "mean.cuh"
#include "cuda_utils/hip/hip_runtime_api.h"

using namespace attribute;

using namespace std;
using namespace volume;



__global__ void _varianceKernel(const Volume* volume, float *data, int texSide) {
    __shared__ unsigned histogram[256];
    __shared__ float buffer[256];
    __shared__ int nVoxels;

    unsigned tid = ThreadIndexing::globalIdx() / ThreadIndexing::totalBlockDim();

    if(tid < volume->voxelCount()) {
        _calcHistogram(volume, (int)tid, texSide, histogram, &nVoxels);
        _calcMean(histogram, nVoxels, buffer);
        _calcVariance(histogram, nVoxels, buffer, buffer[0]);

        if(ThreadIndexing::totalBlockIdx() == 0){
            *(data + tid) = buffer[0];
        }
    }
}


__device__ void _calcVariance(const unsigned *histogram, unsigned nVoxels, float *buffer, float mean) {
    unsigned tid = ThreadIndexing::totalBlockIdx();

    if(tid < ThreadIndexing::totalBlockDim()){
        buffer[tid] = (tid * (float) histogram[tid] - mean) * (tid * (float) histogram[tid] - mean);

        __syncthreads();

        unsigned n = ThreadIndexing::totalBlockDim();
        unsigned halfN;

        while(n > 1) {
            halfN = n / 2;

            if (tid < halfN){
                buffer[tid] = buffer[tid] + buffer[tid + halfN];
            }
            __syncthreads();

            n = halfN;
        }
        if(tid == 0) {
            buffer[tid] = buffer[tid] / nVoxels;
        }
    }

    __syncthreads();
}





Variance::Variance() : Attribute(){
    setType(AttributeType::VARIANCE);
    setDescription("Variance");
}



Variance::~Variance() {

}


thrust::host_vector<float>* Variance::kernelLaunch(const Volume* volume, size_t threadCount, dim3 blockDim, int texSide) {
    thrust::host_vector<float>* hostVector = new thrust::host_vector<float>(threadCount, 0);
    thrust::device_vector<float> devVector = *hostVector;

    blockDim = dim3(256, 1, 1);
    dim3 gridDim = KernelDimsCalculator::instance().calc(blockDim, threadCount * 256);

    _varianceKernel<<<gridDim, blockDim>>>(volume, thrust::raw_pointer_cast(&devVector[0]), texSide);
    getLastCudaError("Mean kernel failed.");
    checkCudaErrors(hipDeviceSynchronize());

    *hostVector = devVector;

    return hostVector;
}
