#include "hip/hip_runtime.h"
#include "glpboimage.h"

#include <cmath>


using namespace std;
using namespace rendering::image;





__host__ __device__ GlPBOImage::GlPBOImage() {
    m_size.x = 0;
    m_size.y = 0;
    m_pixelAspect = 1;
    m_pboData = nullptr;
}





__host__ __device__ GlPBOImage::GlPBOImage(unsigned width, unsigned height) {
    m_size.x = width;
    m_size.y = height;
    m_pixelAspect = 1;
    m_pboData = nullptr;
}





__host__ __device__ GlPBOImage::GlPBOImage(const uint2& size) {
    m_size.x = size.x;
    m_size.y = size.y;
    m_pixelAspect = 1;
    m_pboData = nullptr;
}




__host__ __device__ GlPBOImage::~GlPBOImage() {

}





__host__ __device__ void GlPBOImage::setSize(unsigned width, unsigned height) {
    m_size.x = width;
    m_size.y = height;
}





__host__ __device__ void GlPBOImage::setSize(const uint2& size) {
    m_size = size;
}





__host__ __device__ const uint2& GlPBOImage::size() const {
    return m_size;
}





__host__ __device__ unsigned GlPBOImage::pixelCount() const {
    return m_size.x * m_size.y;
}





__host__ __device__ float GlPBOImage::pixelAspect() const {
    return m_pixelAspect;
}





__host__ __device__ void GlPBOImage::resizePixelAspect(float diagonal, float zoomFunction) {
    m_pixelAspect = (diagonal * zoomFunction) / fminf(m_size.x, m_size.y);
}





__host__ __device__ void GlPBOImage::setPixelColor(int x, int y, const uchar4 &color) {
    int globalIndex = int2ToIntIndex(x, y);

    *(m_pboData + globalIndex * 4 + 0) = static_cast<byte>(color.x);
    *(m_pboData + globalIndex * 4 + 1) = static_cast<byte>(color.y);
    *(m_pboData + globalIndex * 4 + 2) = static_cast<byte>(color.z);
    *(m_pboData + globalIndex * 4 + 3) = static_cast<byte>(color.w);
}





__host__ __device__ void GlPBOImage::setPixelColor(const int2& index, const uchar4 &color) {
    int globalIndex = int2ToIntIndex(index);

    *(m_pboData + globalIndex * 4 + 0) = static_cast<byte>(color.x);
    *(m_pboData + globalIndex * 4 + 1) = static_cast<byte>(color.y);
    *(m_pboData + globalIndex * 4 + 2) = static_cast<byte>(color.z);
    *(m_pboData + globalIndex * 4 + 3) = static_cast<byte>(color.w);
}





__host__ __device__ int GlPBOImage::int2ToIntIndex(int x, int y) const {
    return y * static_cast<int>(m_size.x) + x;
}





__host__ __device__ int GlPBOImage::int2ToIntIndex(const int2& index) const {
    return int2ToIntIndex(index.x, index.y);
}
