#include "hip/hip_runtime.h"
#include "laplacian.cuh"

#include <cmath>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include "cuda_utils/kerneldimscalculator.h"
#include "cuda_utils/threadindexing.cuh"
#include "cuda_utils/hip/hip_runtime_api.h"
#include "quaternion/quaternioncalculator.cuh"



using namespace attribute;

using namespace std;
using namespace volume;
using namespace quaternion;


__global__ void _laplacianKernel(const Volume* volume, float *data) {
    unsigned tid = ThreadIndexing::globalIdx();

    if(tid < volume->voxelCount()){
        *(data + tid) = _calcLaplacian(volume, (int) tid);
    }
}


__device__ float _calcLaplacian(const Volume* volume, int index){
    int3 index3 = volume->indexToIndex3(index);
    float3 laplacian;


    if(index3.x > 0){
        if(static_cast<unsigned>(index3.x + 1) < volume->dim().x){
            //Diferença centrada
            laplacian.x = static_cast<float>(volume->sampleAt(index3.x + 1, index3.y, index3.z))
                        - 2.0f * static_cast<float>(volume->sampleAt(index3.x, index3.y, index3.z))
                        + static_cast<float>(volume->sampleAt(index3.x - 1, index3.y, index3.z));

            laplacian.x = laplacian.x / (volume->spacing().x * volume->spacing().x);
        }
        else{
            //Diferença regressiva
            laplacian.x = static_cast<float>(volume->sampleAt(index3.x, index3.y, index3.z))
                      - 2.0f * static_cast<float>(volume->sampleAt(index3.x - 1, index3.y, index3.z))
                      + static_cast<float>(volume->sampleAt(index3.x - 2, index3.y, index3.z));
            laplacian.x = laplacian.x / (volume->spacing().x * volume->spacing().x);
        }
    }
    else{
        //Diferença progressiva
        laplacian.x = static_cast<float>(volume->sampleAt(index3.x + 2, index3.y, index3.z))
                    - 2.0f * static_cast<float>(volume->sampleAt(index3.x + 1, index3.y, index3.z))
                    + static_cast<float>(volume->sampleAt(index3.x, index3.y, index3.z));
        laplacian.x = laplacian.x / (volume->spacing().x * volume->spacing().x);
    }


    if(index3.y > 0){
        if(static_cast<unsigned>(index3.y + 1) < volume->dim().y){
            //Diferença centrada
            laplacian.y = static_cast<float>(volume->sampleAt(index3.x, index3.y + 1, index3.z))
                      - 2.0f * static_cast<float>(volume->sampleAt(index3.x, index3.y, index3.z))
                      + static_cast<float>(volume->sampleAt(index3.x, index3.y - 1, index3.z));
            laplacian.y = laplacian.y / (volume->spacing().y * volume->spacing().y);
        }
        else {
            //Diferença regressiva
            laplacian.y = static_cast<float>(volume->sampleAt(index3.x, index3.y, index3.z))
                      - 2.0f * static_cast<float>(volume->sampleAt(index3.x, index3.y - 1, index3.z))
                      + static_cast<float>(volume->sampleAt(index3.x, index3.y - 2, index3.z));
            laplacian.y = laplacian.y / (volume->spacing().y * volume->spacing().y);
        }
    }
    else {
        //Diferença progressiva
        laplacian.y = static_cast<float>(volume->sampleAt(index3.x, index3.y + 2, index3.z))
                  - 2.0f * static_cast<float>(volume->sampleAt(index3.x, index3.y + 1, index3.z))
                  + static_cast<float>(volume->sampleAt(index3.x, index3.y, index3.z));
        laplacian.y = laplacian.y / (volume->spacing().y * volume->spacing().y);
    }


    if(index3.z > 0){
        if(static_cast<unsigned>(index3.z + 1) < volume->dim().z){
            //Diferença centrada
            laplacian.z = static_cast<float>(volume->sampleAt(index3.x, index3.y, index3.z + 1))
                      - 2.0f * static_cast<float>(volume->sampleAt(index3.x, index3.y, index3.z))
                      + static_cast<float>(volume->sampleAt(index3.x, index3.y, index3.z - 1));
            laplacian.z = laplacian.z / (volume->spacing().z * volume->spacing().z);
        }
        else{
            //Diferença regressiva
            laplacian.z = static_cast<float>(volume->sampleAt(index3.x, index3.y, index3.z))
                      - 2.0f * static_cast<float>(volume->sampleAt(index3.x, index3.y, index3.z - 1))
                      + static_cast<float>(volume->sampleAt(index3.x, index3.y, index3.z - 2));
            laplacian.z = laplacian.z / (volume->spacing().z * volume->spacing().z);
        }
    }
    else {
        //Diferença progressiva
        laplacian.z = static_cast<float>(volume->sampleAt(index3.x, index3.y, index3.z + 2))
                  - 2.0f * static_cast<float>(volume->sampleAt(index3.x, index3.y, index3.z + 1))
                  + static_cast<float>(volume->sampleAt(index3.x, index3.y, index3.z));
        laplacian.z = laplacian.z / (volume->spacing().z * volume->spacing().z);
    }

    return QuaternionCalculator::norm(laplacian);
}



Laplacian::Laplacian() : Attribute(){
    setType(AttributeType::LAPLACIAN);
    setName("Laplacian");
}



Laplacian::~Laplacian() {

}


thrust::host_vector<float>* Laplacian::kernelLaunch(const Volume* volume, size_t threadCount, dim3 blockDim, int mask) {
    thrust::host_vector<float>* hostVector = new thrust::host_vector<float>(threadCount, 0);
    thrust::device_vector<float> devVector = *hostVector;

    dim3 gridDim = KernelDimsCalculator::instance().calc(blockDim, threadCount);

    _laplacianKernel<<<gridDim, blockDim>>>(volume, thrust::raw_pointer_cast(&devVector[0]));
    getLastCudaError("Laplacian kernel failed.");
    checkCudaErrors(hipDeviceSynchronize());

    *hostVector = devVector;

    return hostVector;
}
