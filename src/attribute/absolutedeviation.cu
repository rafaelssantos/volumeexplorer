#include "hip/hip_runtime.h"
#include "absolutedeviation.cuh"

#include <cmath>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include "cuda_utils/kerneldimscalculator.h"
#include "cuda_utils/threadindexing.cuh"
#include "mean.cuh"
#include "cuda_utils/hip/hip_runtime_api.h"


using namespace attribute;

using namespace std;
using namespace volume;





__global__ void _absoluteDeviationKernel(const Volume* volume, float *data, int texSide) {
	__shared__ unsigned histogram[256];
	__shared__ float buffer[256];
    __shared__ int nVoxels;

	unsigned index = ThreadIndexing::globalIdx() / ThreadIndexing::totalBlockDim();

	if(index < volume->voxelCount()) {
        _calcHistogram(volume, (int)index, texSide, histogram, &nVoxels);
        _calcMean(histogram, nVoxels, buffer);
        _calcAbsoluteDeviation(histogram, nVoxels, buffer, buffer[0]);

		if(ThreadIndexing::totalBlockIdx() == 0){
			*(data + index) = buffer[0];
		}
	}
}





__device__ void _calcAbsoluteDeviation(const unsigned *histogram, unsigned nVoxels, float *buffer, float mean) {
    unsigned tid = ThreadIndexing::totalBlockIdx();

    if(tid < ThreadIndexing::totalBlockDim()){
        buffer[tid] = fabs(tid * (float) histogram[tid] - mean);

        __syncthreads();

        unsigned n = ThreadIndexing::totalBlockDim();
        unsigned halfN;

        while(n > 1) {
            halfN = n / 2;

            if (tid < halfN){
                buffer[tid] = buffer[tid] + buffer[tid + halfN];
            }
            __syncthreads();

            n = halfN;
        }
        if(tid == 0) {
            buffer[tid] = buffer[tid] / nVoxels;
        }
    }

	__syncthreads();
}





AbsoluteDeviation::AbsoluteDeviation() : Attribute(){
	setType(AttributeType::ABSOLUTE_DEVIATION);
	setName("Absolute Deviation");
}





AbsoluteDeviation::~AbsoluteDeviation() {

}





thrust::host_vector<float>* AbsoluteDeviation::kernelLaunch(const Volume* volume, size_t threadCount, dim3 blockDim, int texSide) {
    thrust::host_vector<float>* hostVector = new thrust::host_vector<float>(threadCount, 0);
    thrust::device_vector<float> devVector = *hostVector;

	blockDim = dim3(256, 1, 1);
    dim3 gridDim = KernelDimsCalculator::instance().calc(blockDim, threadCount * 256);

	_absoluteDeviationKernel<<<gridDim, blockDim>>>(volume, thrust::raw_pointer_cast(&devVector[0]), texSide);
    getLastCudaError("Absolute deviation kernel failed.");
	checkCudaErrors(hipDeviceSynchronize());

	*hostVector = devVector;

	return hostVector;
}
