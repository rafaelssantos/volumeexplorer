#include "hip/hip_runtime.h"
#include "standarddeviation.cuh"


#include <cmath>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include "cuda_utils/kerneldimscalculator.h"
#include "cuda_utils/threadindexing.cuh"
#include "mean.cuh"
#include "cuda_utils/hip/hip_runtime_api.h"

using namespace attribute;

using namespace std;
using namespace volume;



__global__ void _standardDeviationKernel(const Volume* volume, float *data, int texSide) {
    __shared__ unsigned histogram[256];
    __shared__ float buffer[256];
    __shared__ int nVoxels;

    unsigned tid = ThreadIndexing::globalIdx() / ThreadIndexing::totalBlockDim();

    if(tid < volume->voxelCount()) {
        _calcHistogram(volume, (int)tid, texSide, histogram, &nVoxels);
        _calcMean(histogram, nVoxels, buffer);
        _calcStandardDeviation(histogram, nVoxels, buffer, buffer[0]);

        if(ThreadIndexing::totalBlockIdx() == 0){
            *(data + tid) = buffer[0];
        }
    }
}


__device__ void _calcStandardDeviation(const unsigned *histogram, int nVoxels, float *buffer, float mean) {
    unsigned tid = ThreadIndexing::totalBlockIdx();

    if(tid < ThreadIndexing::totalBlockDim()){
        buffer[tid] = (tid * (float) histogram[tid] - mean) * (tid * (float) histogram[tid] - mean);

        __syncthreads();

        unsigned n = ThreadIndexing::totalBlockDim();
        unsigned halfN;

        while(n > 1) {
            halfN = n / 2;

            if (tid < halfN){
                buffer[tid] = buffer[tid] + buffer[tid + halfN];
            }
            __syncthreads();

            n = halfN;
        }
        if(tid == 0) {
            buffer[tid] = sqrtf(buffer[tid] / nVoxels);
        }
    }

    __syncthreads();
}





StandardDeviation::StandardDeviation() : Attribute(){
    setType(AttributeType::STANDARD_DEVIATION);
    setName("Standard Deviation");
}



StandardDeviation::~StandardDeviation() {

}


thrust::host_vector<float>* StandardDeviation::kernelLaunch(const Volume* volume, size_t threadCount, dim3 blockDim, int texSide) {
    thrust::host_vector<float>* hostVector = new thrust::host_vector<float>(threadCount, 0);
    thrust::device_vector<float> devVector = *hostVector;

    blockDim = dim3(256, 1, 1);
    dim3 gridDim = KernelDimsCalculator::instance().calc(blockDim, threadCount * 256);

    _standardDeviationKernel<<<gridDim, blockDim>>>(volume, thrust::raw_pointer_cast(&devVector[0]), texSide);
    getLastCudaError("Standard deviation kernel failed.");
    checkCudaErrors(hipDeviceSynchronize());

    *hostVector = devVector;

    return hostVector;
}
