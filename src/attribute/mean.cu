#include "hip/hip_runtime.h"
#include "mean.cuh"

#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include "cuda_utils/kerneldimscalculator.h"
#include "cuda_utils/threadindexing.cuh"
#include "cuda_utils/hip/hip_runtime_api.h"

using namespace attribute;

using namespace std;
using namespace volume;



__global__ void _meanKernel(const Volume* volume, float *data, int texSide) {
	__shared__ unsigned histogram[256];
	__shared__ float buffer[256];
    __shared__ int nVoxels;

    unsigned tid = ThreadIndexing::globalIdx() / ThreadIndexing::totalBlockDim();

    if(tid < volume->voxelCount()) {
        _calcHistogram(volume, (int)tid, texSide, histogram, &nVoxels);
        _calcMean(histogram, nVoxels, buffer);

		if(ThreadIndexing::totalBlockIdx() == 0){
            *(data + tid) = buffer[0];
		}
	}
}


__device__ void _calcMean(const unsigned *histogram, unsigned nVoxels, float *buffer) {
    unsigned tid = ThreadIndexing::totalBlockIdx();

    if(tid < ThreadIndexing::totalBlockDim()){
        buffer[tid] = tid * (float) histogram[tid];

		__syncthreads();

        unsigned n = ThreadIndexing::totalBlockDim();
        unsigned halfN;

        while(n > 1) {
            halfN = n / 2;

            if (tid < halfN){
                buffer[tid] = buffer[tid] + buffer[tid + halfN];

			}
			__syncthreads();

            n = halfN;
		}
        if(tid == 0){
            buffer[0] = buffer[0] / nVoxels;
        }
	}

	__syncthreads();
}


__device__ void _calcHistogram(const Volume *volume, int index, int texSide, unsigned* histogram, int* nVoxels) {
    unsigned tid = ThreadIndexing::totalBlockIdx();

    if(tid == 0){
        *nVoxels = 0;
	}

    if(tid < 256){
        histogram[tid] = 0;
	}
	__syncthreads();

    int texDim3 = texSide * texSide * texSide;

    int3 volIndex3;
    int3 texIndex3;

    while(tid < texDim3) {
        texIndex3.z = tid / (texSide * texSide);
        texIndex3.y = (tid % (texSide * texSide)) / texSide;
        texIndex3.x = (tid % (texSide * texSide)) % texSide;

        volIndex3 = volume->indexToIndex3(index);
        volIndex3.x = volIndex3.x - texSide / 2 + texIndex3.x;
        volIndex3.y = volIndex3.y - texSide / 2 + texIndex3.y;
        volIndex3.z = volIndex3.z - texSide / 2 + texIndex3.z;

        if(volume->isIndex3In(volIndex3)) {
            atomicAdd((histogram + (int) volume->sampleAt(volIndex3)), 1);
            atomicAdd(nVoxels, 1);
		}

        tid = tid + ThreadIndexing::totalBlockDim();
	}

	__syncthreads();
}




Mean::Mean() : Attribute(){
	setType(AttributeType::MEAN);
	setName("Mean");
}



Mean::~Mean() {

}


thrust::host_vector<float>* Mean::kernelLaunch(const Volume* volume, size_t threadCount, dim3 blockDim, int mask) {
    thrust::host_vector<float>* hostVector = new thrust::host_vector<float>(threadCount, 0);
    thrust::device_vector<float> devVector = *hostVector;

	blockDim = dim3(256, 1, 1);
    dim3 gridDim = KernelDimsCalculator::instance().calc(blockDim, threadCount * 256);

	_meanKernel<<<gridDim, blockDim>>>(volume, thrust::raw_pointer_cast(&devVector[0]), mask);
	getLastCudaError("Mean kernel failed.");
	checkCudaErrors(hipDeviceSynchronize());

	*hostVector = devVector;

	return hostVector;
}
