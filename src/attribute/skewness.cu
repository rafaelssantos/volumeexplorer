#include "hip/hip_runtime.h"
#include "skewness.cuh"


#include <cmath>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include "cuda_utils/kerneldimscalculator.h"
#include "cuda_utils/threadindexing.cuh"
#include "cuda_utils/hip/hip_runtime_api.h"
#include "mean.cuh"
#include "standarddeviation.cuh"

using namespace attribute;

using namespace std;
using namespace volume;



__global__ void _skewnessKernel(const Volume* volume, float *data, int texSide) {
    __shared__ unsigned histogram[256];
    __shared__ float buffer[256];
    __shared__ int nVoxels;

    unsigned tid = ThreadIndexing::globalIdx() / ThreadIndexing::totalBlockDim();

    if(tid < volume->voxelCount()) {
        _calcHistogram(volume, (int)tid, texSide, histogram, &nVoxels);
        _calcMean(histogram, nVoxels, buffer);
        float mean = buffer[0];

        _calcStandardDeviation(histogram, nVoxels, buffer, mean);
        float standardDeviation = buffer[0];

        _calcSkewness(histogram, nVoxels, buffer, mean, standardDeviation);

        if(ThreadIndexing::totalBlockIdx() == 0){
            *(data + tid) = buffer[0];
        }
    }
}


__device__ void _calcSkewness(const unsigned *histogram, unsigned nVoxels, float *buffer, float mean, float standadDeviation) {
    unsigned tid = ThreadIndexing::totalBlockIdx();

    if(tid < ThreadIndexing::totalBlockDim()){
        float value = tid  - mean;
        buffer[tid] =  value * value * value * (float) histogram[tid];

        __syncthreads();

        unsigned n = ThreadIndexing::totalBlockDim();
        unsigned halfN;

        while(n > 1) {
            halfN = n / 2;

            if (tid < halfN){
                 buffer[tid] = buffer[tid] + buffer[tid + halfN];
            }
            __syncthreads();

            n = halfN;
        }

        if(tid == 0){
            if(standadDeviation != 0) {
                buffer[tid] = buffer[tid] / nVoxels;
                buffer[tid] = buffer[tid] / (standadDeviation * standadDeviation * standadDeviation);
            }
            else{
                buffer[tid] = 0;
            }
        }
    }

    __syncthreads();
}





Skewness::Skewness() : Attribute(){
    setType(AttributeType::SKEWNESS);
    setDescription("Skewness");
}



Skewness::~Skewness() {

}


thrust::host_vector<float>* Skewness::kernelLaunch(const Volume* volume, size_t threadCount, dim3 blockDim, int texSide) {
    thrust::host_vector<float>* hostVector = new thrust::host_vector<float>(threadCount, 0);
    thrust::device_vector<float> devVector = *hostVector;

    blockDim = dim3(256, 1, 1);
    dim3 gridDim = KernelDimsCalculator::instance().calc(blockDim, threadCount * 256);

    _skewnessKernel<<<gridDim, blockDim>>>(volume, thrust::raw_pointer_cast(&devVector[0]), texSide);
    getLastCudaError("Skewness kernel failed.");
    checkCudaErrors(hipDeviceSynchronize());

    *hostVector = devVector;

    return hostVector;
}
