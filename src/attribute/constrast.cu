#include "hip/hip_runtime.h"
#include "constrast.cuh"


#include <cmath>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include "entropy.cuh"
#include "cuda_utils/kerneldimscalculator.h"
#include "cuda_utils/threadindexing.cuh"
#include "cuda_utils/hip/hip_runtime_api.h"
#include "mean.cuh"

using namespace attribute;

using namespace std;
using namespace volume;



__global__ void _constrastKernel(const Volume* volume, float *data, int texSide) {
    __shared__ unsigned histogram[256];
    __shared__ float buffer[256];
    __shared__ int nVoxels;

    unsigned index = ThreadIndexing::globalIdx() / ThreadIndexing::totalBlockDim();

    if(index < volume->voxelCount()) {
        _calcHistogram(volume, (int)index, texSide, histogram, &nVoxels);

        _calcContrast(histogram, nVoxels, buffer);

        if(ThreadIndexing::totalBlockIdx() == 0){
            *(data + index) = buffer[0];
        }
    }
}


__device__ void _calcContrast(const unsigned *histogram, unsigned nVoxels, float *buffer) {
    unsigned tid = ThreadIndexing::totalBlockIdx();

    if(tid < ThreadIndexing::totalBlockDim()){
        buffer[tid] = (tid * tid) *((float) histogram[tid] / (float) nVoxels);

        __syncthreads();

        unsigned n = ThreadIndexing::totalBlockDim();
        unsigned halfN;

        while(n > 1) {
            halfN = n / 2;

            if (tid < halfN){
                buffer[tid] = buffer[tid] + buffer[tid + halfN];
            }
            __syncthreads();

            n = halfN;
        }
        if(tid == 0){
            buffer[tid] = - buffer[tid];
        }
    }

    __syncthreads();
}





Contrast::Contrast() : Attribute(){
    setType(AttributeType::CONTRAST);
    setName("Sum Entropy");
}



Contrast::~Contrast() {

}


thrust::host_vector<float>* Contrast::kernelLaunch(const Volume* volume, size_t threadCount, dim3 blockDim, int texSide) {
    thrust::host_vector<float>* hostData = new thrust::host_vector<float>(threadCount, 0);
    thrust::device_vector<float> devData = *hostData;

    blockDim = dim3(256, 1, 1);
    dim3 gridDim = KernelDimsCalculator::instance().calc(blockDim, threadCount * 256);

    _constrastKernel<<<gridDim, blockDim>>>(volume, thrust::raw_pointer_cast(&devData[0]), texSide);
    getLastCudaError("Constrast kernel failed.");
    checkCudaErrors(hipDeviceSynchronize());

    *hostData = devData;

    return hostData;
}
