#include "hip/hip_runtime.h"
#include "intensity.cuh"

#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include "cuda_utils/kerneldimscalculator.h"
#include "cuda_utils/threadindexing.cuh"
#include "cuda_utils/hip/hip_runtime_api.h"


using namespace attribute;

using namespace std;
using namespace volume;





__global__ void _instensityKernel(const Volume* volume, float *data) {
	unsigned id = ThreadIndexing::globalIdx();

	if(id < volume->voxelCount()){
        *(data + id) = Intensity::_calcIntensity(volume, (int) id);
	}
}


__device__ float Intensity::_calcIntensity(const Volume* volume, int index){
    return static_cast<float>(volume->sampleAt(volume->indexToIndex3(index)));
}



Intensity::Intensity() : Attribute(){
	setType(AttributeType::INTENSITY);
	setName("Intesity");
}



Intensity::~Intensity() {

}


thrust::host_vector<float>* Intensity::kernelLaunch(const Volume* volume, size_t threadCount, dim3 blockDim, int) {
	thrust::host_vector<float>* hostVector = new thrust::host_vector<float>(threadCount, 0);
	thrust::device_vector<float> devVector = *hostVector;

    dim3 gridDim = KernelDimsCalculator::instance().calc(blockDim, threadCount);

	_instensityKernel<<<gridDim, blockDim>>>(volume, thrust::raw_pointer_cast(&devVector[0]));
    getLastCudaError("Intensity kernel failed.");
	checkCudaErrors(hipDeviceSynchronize());

	*hostVector = devVector;

	return hostVector;
}
