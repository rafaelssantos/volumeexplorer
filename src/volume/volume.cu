#include "hip/hip_runtime.h"
#include "volume.cuh"

#include <cmath>
#include <string>



using namespace std;
using namespace volume;
using namespace grid;





__host__ __device__ Volume::Volume(unsigned x, unsigned y, unsigned z) : Grid3D(x, y, z) {
    m_data = nullptr;
}





__host__ __device__ Volume::Volume(const uint3& dim) : Grid3D(dim) {
    m_data = nullptr;
}





__host__ __device__ Volume::~Volume() {
#ifndef __CUDA_ARCH__
    delete []  m_data;
    m_data = nullptr;
#endif
}





__host__ __device__ std::byte Volume::sampleAt(int x, int y, int z) const {
    return *(m_data + index3toIndex(x, y, z));
}





__host__ __device__ std::byte Volume::sampleAt(const int3& index) const {
    return *(m_data + index3toIndex(index));
}



__host__ __device__ bool Volume::isInCircumsphere(float x, float y, float z) const {
    float opCatheter;
    float adjCatheter;
    float ray;
    float posToCenter;

    opCatheter = diameter()/2.0f;
    adjCatheter = diameter();
    ray = sqrtf(opCatheter * opCatheter + adjCatheter * adjCatheter);

    float dist_x = (x - center().x);
    float dist_y = (y - center().y);
    float dist_z = (z - center().z);

    posToCenter = sqrtf(dist_x * dist_x + dist_y * dist_y + dist_z * dist_z);

    return posToCenter <= ray;
}





__host__ __device__ bool Volume::isInCircumsphere(const float3& pos) const {
    float opCatheter;
    float adjCatheter;
    float ray;
    float posToCenter;

    opCatheter = diameter() / 2.0f;
    adjCatheter = diameter();
    ray = sqrtf(opCatheter * opCatheter + adjCatheter * adjCatheter);

    float dist_x = (pos.x - center().x);
    float dist_y = (pos.y - center().y);
    float dist_z = (pos.z - center().z);

    posToCenter = sqrtf(dist_x * dist_x + dist_y * dist_y + dist_z * dist_z);

    return posToCenter <= ray;
}



std::string Volume::toString() {
    return Grid3D::toString() + " | Voxels: " + to_string(voxelCount());
}
