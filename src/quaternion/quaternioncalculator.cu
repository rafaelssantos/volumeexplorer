#include "hip/hip_runtime.h"
#include "quaternioncalculator.cuh"



using namespace quaternion;




Quaternion QuaternionCalculator::sum(const Quaternion& q1, const Quaternion& q2) {
    Quaternion qr;

    qr.s = q1.s + q2.s;
    qr.x = q1.x + q2.x;
    qr.y = q1.y + q2.y;
    qr.z = q1.z + q2.z;

    return qr;
}






__host__ __device__  Quaternion QuaternionCalculator::sub(const Quaternion& q1, const Quaternion& q2) {
    Quaternion qr;

    qr.s = q1.s - q2.s;
    qr.x = q1.x - q2.x;
    qr.y = q1.y - q2.y;
    qr.z = q1.z - q2.z;

    return qr;
}






__host__ __device__ Quaternion QuaternionCalculator::scale(float s, const Quaternion& q) {
    Quaternion qr;

    qr.s = s * q.s;
    qr.x = s * q.x;
    qr.y = s * q.y;
    qr.z = s * q.z;

    return qr;
}


__host__ __device__ float3 QuaternionCalculator::scale(float s, const float3& p) {
    float3 pr;

    pr.x = s * p.x;
    pr.y = s * p.y;
    pr.z = s * p.z;

    return pr;
}







__host__ __device__ Quaternion QuaternionCalculator::crossProduct(Quaternion q1, Quaternion q2) {
    Quaternion qr;

    qr.s = q1.s * q2.s - q1.x * q2.x - q1.y * q2.y - q1.z * q2.z;
    qr.x = q1.s * q2.x + q2.s * q1.x + q1.y * q2.z - q1.z * q2.y;
    qr.y = q1.s * q2.y + q2.s * q1.y + q1.z * q2.x - q1.x * q2.z;
    qr.z = q1.s * q2.z + q2.s * q1.z + q1.x * q2.y - q1.y * q2.x;

    return qr;
}






__host__ __device__ Quaternion QuaternionCalculator::conjugation(const Quaternion& q) {
    Quaternion qr;

    qr.s = q.s;
    qr.x = -q.x;
    qr.y = -q.y;
    qr.z = -q.z;

    return qr;
}






__host__ __device__ float QuaternionCalculator::norm(const Quaternion& q) {
    return sqrtf(q.s * q.s + q.x * q.x + q.y * q.y + q.z * q.z);
}




__host__ __device__ float QuaternionCalculator::norm(const float3& pos) {
    return sqrtf(pos.x * pos.x + pos.y * pos.y + pos.z * pos.z);
}




__host__ __device__ Quaternion QuaternionCalculator::normalize(const Quaternion& q) {
    float norm = QuaternionCalculator::norm(q);

    if (norm == 0.0f) {
        return q;
    }

    return scale(1.0f / norm, q);
}





__host__ __device__ float3 QuaternionCalculator::normalize(const float3& p) {
    float norm = QuaternionCalculator::norm(p);

    if (norm == 0.0f) {
        return p;
    }

    return scale(1.0f / norm, p);
}






__host__ __device__ Quaternion QuaternionCalculator::multiplicativeInverse(const Quaternion& q) {
    float norm = QuaternionCalculator::norm(q);

    if (norm  == 0.0f) {
        return q;
    }
    else{
        return scale(1.0f / norm, conjugation(q));
    }
}






__host__ __device__ Quaternion QuaternionCalculator::linearInterpolation(const Quaternion& q1, const Quaternion& q2, float t) {
    return sum(scale((1.0f - t), q1), scale(t, q2));
}






__host__ __device__ float QuaternionCalculator::dotProduct(const Quaternion& q1, const Quaternion& q2) {
    return sqrtf(q1.s * q2.s + q1.x * q2.x + q1.y * q2.y + q1.z * q2.z);
}






__host__ __device__ float QuaternionCalculator::dotProduct(const float3& p1, const float3& p2) {
    return p1.x * p2.x + p1.y * p2.y + p1.z * p2.z;
}






__host__ __device__ Quaternion QuaternionCalculator::angularInterpolation(const Quaternion& q1, const Quaternion& q2, float u) {
    const float HALF_PI = 1.57079632679489661923132169164f;
    const float EPSILON = 0.000001f;
    float cos_omega;
    float sin_omega;
    float omega;
    float s1, s2;

    Quaternion q2Clone = q2; //Cópia devido ao const

    if (norm(sub(q1, q2Clone)) > norm(sum(q1, q2Clone))) {
        q2Clone = scale(-1, q2Clone);
    }

    cos_omega = dotProduct(q1, q2Clone);

    if ((1.0f - cos_omega) < EPSILON) {
        return linearInterpolation(q1, q2Clone, u);
    }

    if ((1.0f + cos_omega) < EPSILON) {
        Quaternion q2a;
        q2a.s = -q2Clone.x;
        q2a.x = q2Clone.s;
        q2a.y = -q2Clone.z;
        q2a.z = q2Clone.y;
        s1 = sinf((1.0f - u) * HALF_PI);
        s2 = sinf(u * HALF_PI);
        return sum(scale(s1, q1), scale(s2, q2a));
    }
    omega = acosf(cos_omega);
    sin_omega = sinf(omega);
    s1 = sinf((1.0f - u) * omega) / sin_omega;
    s2 = sinf(u * omega) / sin_omega;

    return sum(scale(s1, q1), scale(s2, q2Clone));
}






__host__ __device__ RotQuaternion QuaternionCalculator::toRotation(const Quaternion& q) {
    const float halfpi = 1.57079632679489661923132169164f;

    RotQuaternion rr;
    rr.theta = (acosf(q.s) * 180.0f) / halfpi;
    rr.nx = q.x;
    rr.ny = q.y;
    rr.nz = q.z;

    return rr;
}






__host__ __device__ Quaternion QuaternionCalculator::fromRotation(const RotQuaternion& r) {
    const float HALF_PI = 1.57079632679489661923132169164f;
    const float EPSILON = 0.000001f;
    float halftheta;
    float sinhalftheta;
    float axisnorm;
    Quaternion qr;

    if ((axisnorm = sqrtf(r.nx * r.nx + r.ny * r.ny + r.nz * r.nz)) < EPSILON) {
        qr.s = 1.0;
        qr.x = 0.0;
        qr.y = 0.0;
        qr.z = 0.0;
        return qr;
    }

    halftheta = (r.theta * HALF_PI) / 180.0f;
    sinhalftheta = sinf(halftheta);
    qr.s = cosf(halftheta);
    qr.x = sinhalftheta * (r.nx / axisnorm);
    qr.y = sinhalftheta * (r.ny / axisnorm);
    qr.z = sinhalftheta * (r.nz / axisnorm);

    return qr;
}






__host__ __device__ float3 QuaternionCalculator::toPosition(const Quaternion& q) {
    float3 pr;

    pr.x = q.x;
    pr.y = q.y;
    pr.z = q.z;

    return pr;
}






__host__ __device__ Quaternion QuaternionCalculator::fromPosition(const float3& p) {
    Quaternion qr;

    qr.s = 0.0;
    qr.x = p.x;
    qr.y = p.y;
    qr.z = p.z;

    return qr;
}






__host__ __device__ RotQuaternion QuaternionCalculator::compose(const RotQuaternion& r1, const RotQuaternion& r2) {
    return toRotation(crossProduct(fromRotation(r2), fromRotation(r1)));
}






__host__ __device__ float3 QuaternionCalculator::rotate(const float3& p, const RotQuaternion& r) {
    return toPosition(crossProduct(fromRotation(r), crossProduct(fromPosition(p), conjugation(fromRotation(r)))));
}






 __host__ __device__ float3 QuaternionCalculator::rotate(const float3& p, const RotQuaternion& r, const float3& rotCenter) {
    float3 pClone = p;

    pClone.x -= rotCenter.x;
    pClone.y -= rotCenter.y;
    pClone.z -= rotCenter.z;

    pClone = QuaternionCalculator::rotate(pClone, r);

    pClone.x += rotCenter.x;
    pClone.y += rotCenter.y;
    pClone.z += rotCenter.z;

    return pClone;
}






__host__ __device__ Quaternion QuaternionCalculator::crossProductWithoutValue(const Quaternion& q1, const Quaternion& q2) {
    Quaternion r;

    r.x = q1.y * q2.z - q1.z * q2.y;
    r.y = q1.x * q2.z - q1.z * q2.x;
    r.z = q1.x * q2.y - q1.y * q2.x;

    return r;
}







__host__ __device__ RotQuaternion QuaternionCalculator::calcRotation(const float3& p1, const float3& p2) {
    const float HALFPI = 1.57079632679489661923132169164f;

    float3 p1Clone = toPosition(normalize(fromPosition(p1)));
    float3 p2Clone = toPosition(normalize(fromPosition(p2)));
    RotQuaternion rot = QuaternionCalculator::toRotation(normalize(crossProductWithoutValue(fromPosition(p1), fromPosition(p2))));
    rot.theta = 90.0f * acosf(QuaternionCalculator::dotProduct(p1Clone, p2Clone)) / HALFPI;


    return rot;
}






__host__ __device__ float QuaternionCalculator::distance(const float3& p1, const float3& p2) {
    return sqrtf((p1.x - p2.x) * (p1.x - p2.x) + (p1.y - p2.y) * (p1.y - p2.y) + (p1.z - p2.z) * (p1.z - p2.z));
}
