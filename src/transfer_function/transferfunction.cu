#include "hip/hip_runtime.h"
#include "transferfunction.cuh"

#include <cmath>
#include "quaternion/quaternioncalculator.cuh"


using namespace quaternion;
using namespace grid;
using namespace transfer_function;






__host__ __device__ TransferFunction::TransferFunction(unsigned x, unsigned y, unsigned z) : Grid3D(x, y, z) {
    m_colors = nullptr;
    m_normals = nullptr;

    m_bgColor.x = 1.0f;
    m_bgColor.y = 1.0f;
    m_bgColor.z = 1.0f;
    m_bgColor.w = 1.0f;
}





__host__ __device__ TransferFunction::TransferFunction(const uint3& dim) : Grid3D(dim) {
    m_colors = nullptr;
    m_normals = nullptr;

    m_bgColor.x = 1.0f;
    m_bgColor.y = 1.0f;
    m_bgColor.z = 1.0f;
    m_bgColor.w = 1.0f;
}




__host__ __device__ TransferFunction::~TransferFunction() {
#ifndef __CUDA_ARCH__
    delete []  m_colors;
    m_colors = nullptr;

    delete [] m_normals;
    m_normals = nullptr;
#endif
}


__host__ __device__ float4 TransferFunction::colorAt(float x, float y, float z, const float3& lightPos, const Illumination &illumination, const Interpolation &filterType) const {
    float3 pos;
    pos.x = x;
    pos.y = y;
    pos.z = z;

    return colorAt(pos, lightPos, illumination, filterType);
}




__host__ __device__ float4 TransferFunction::colorAt(const float3& pos, const float3& lightPos, const Illumination &illum, const Interpolation &interpSpecs) const {
    float4 color;

    if(interpSpecs == Interpolation::TRILINEAR) {
        color = trilinearColorAt(pos);
    }
    else {
        color = nearestColorAt(pos);
    }

    if (illum.enabled){
        float3 shading = blinnPhongIllum(pos, lightPos, illum);

        color.x = fminf(1.0f, color.x * shading.x + color.x * shading.y + shading.z);
        color.y = fminf(1.0f, color.y * shading.x + color.y * shading.y + shading.z);
        color.z = fminf(1.0f, color.z * shading.x + color.z * shading.y + shading.z);
    }


    return color;
}





__host__ __device__ float4 TransferFunction::sampleAt(int x, int y, int z) const {
    return sampleAt(index3toIndex(x,y,z));
}




__host__ __device__ float4 TransferFunction::sampleAt(const int3& index) const {
    return sampleAt(index3toIndex(index));
}



__host__ __device__ float4 TransferFunction::sampleAt(size_t index) const {
    uchar4 color = *(m_colors + index);
    float4 colorF;


    colorF.x = color.x / 255.0f;
    colorF.y = color.y / 255.0f;
    colorF.z = color.z / 255.0f;
    colorF.w = color.w / 255.0f;

    return colorF;
}



__host__ __device__ float3 TransferFunction::blinnPhongIllum(const float3& pos, const float3& lightPos, const Illumination &illumination) const {
    int3 index3 = roundPosToIndex(pos);

    float3 normal = *(m_normals + index3toIndex(index3));

    float3 lightDir;
    lightDir.x = (lightPos.x - pos.x);
    lightDir.y = (lightPos.y - pos.y);
    lightDir.z = (lightPos.z - pos.z);
    lightDir = QuaternionCalculator::normalize(lightDir);

    float3 halfwayDir;
    halfwayDir.x = lightPos.x + pos.x;
    halfwayDir.y = lightPos.y + pos.y;
    halfwayDir.z = lightPos.z + pos.z;
    halfwayDir= QuaternionCalculator::normalize(halfwayDir);



    float ambient = illumination.ambientK;
    float diffuse = illumination.diffuseK * fmaxf(QuaternionCalculator::dotProduct(lightDir, normal), 0.0f);
    float specular = illumination.specularK * fmaxf(QuaternionCalculator::dotProduct(halfwayDir, normal), 0.0f);



    float3 light;

    light.x = ambient;
    light.y = diffuse;
    light.z = specular;

    return light;

}



__host__ __device__ void TransferFunction::setSampleAt(int x, int y, int z, const uchar4 &value) {
    *(m_colors + index3toIndex(x, y , z)) = value;
}






__host__ __device__ void TransferFunction::setSampleAt(const int3& index, const uchar4 &value) {
    *(m_colors + index3toIndex(index)) = value;
}




__host__ __device__ void TransferFunction::setSampleAt(size_t index, const uchar4 &value) {
    *(m_colors + index) = value;

}






__host__ __device__ const float4& TransferFunction::backgroundColor() const {
    return m_bgColor;
}





__host__ __device__ void TransferFunction::setBackGroundColor(const float4& bgColor) {
    m_bgColor = bgColor;
}






__host__ __device__ void TransferFunction::setBackGroundColor(const uchar4 &bgColor) {
    m_bgColor.x = bgColor.x / 255.0f;
    m_bgColor.y = bgColor.y / 255.0f;
    m_bgColor.z = bgColor.z / 255.0f;
    m_bgColor.w = bgColor.w / 255.0f;
}






__host__ __device__ void TransferFunction::setBackGroundColor(float red, float green, float blue, float opacity) {
    m_bgColor.x = red;
    m_bgColor.y = green;
    m_bgColor.z = blue;
    m_bgColor.w = opacity;
}






__host__ __device__ void TransferFunction::setBackGroundColor(std::byte red, std::byte green, std::byte blue, std::byte opacity) {
    m_bgColor.x = static_cast<unsigned char>(red) / 255.0f;
    m_bgColor.y = static_cast<unsigned char>(green) / 255.0f;
    m_bgColor.z = static_cast<unsigned char>(blue) / 255.0f;
    m_bgColor.w = static_cast<unsigned char>(opacity)/ 255.0f;
}





__host__ __device__ float4 TransferFunction::nearestColorAt(const float3& pos) const {
    return sampleAt(roundPosToIndex(pos));
}





__host__ __device__ float4 TransferFunction::trilinearColorAt(const float3& p) const {
    int3 index0 = floorPosToIndex(p);
    int3 index1 = {index0.x + 1, index0.y + 1, index0.z + 1};

    if(isIndex3In(index0) && isIndex3In(index1)){
        float xD, yD, zD;

        xD = (p.x - index0.x * spacing().x) / spacing().x;
        yD = (p.y - index0.y * spacing().y) / spacing().y;
        zD = (p.z - index0.z * spacing().z) / spacing().z;


        float4 color000, color001, color010, color011, color100, color101, color110, color111;

        color000 = sampleAt(index0.x, index0.y, index0.z);
        color001 = sampleAt(index0.x, index0.y, index1.z);
        color010 = sampleAt(index0.x, index1.y, index0.z);
        color011 = sampleAt(index0.x, index1.y, index1.z);
        color100 = sampleAt(index1.x, index0.y, index0.z);
        color101 = sampleAt(index1.x, index0.y, index1.z);
        color110 = sampleAt(index1.x, index1.y, index0.z);
        color111 = sampleAt(index1.x, index1.y, index1.z);

        float4 color00, color01, color10, color11;

        color00.x = color000.x * (1 - xD) + color100.x * xD;
        color00.y = color000.y * (1 - xD) + color100.y * xD;
        color00.z = color000.z * (1 - xD) + color100.z * xD;
        color00.w = color000.w * (1 - xD) + color100.w * xD;

        color01.x = color001.x * (1 - xD) + color101.x * xD;
        color01.y = color001.y * (1 - xD) + color101.y * xD;
        color01.z = color001.z * (1 - xD) + color101.z * xD;
        color01.w = color001.w * (1 - xD) + color101.w * xD;

        color10.x = color010.x * (1 - xD) + color110.x * xD;
        color10.y = color010.y * (1 - xD) + color110.y * xD;
        color10.z = color010.z * (1 - xD) + color110.z * xD;
        color10.w = color010.w * (1 - xD) + color110.w * xD;

        color11.x = color011.x * (1 - xD) + color111.x * xD;
        color11.y = color011.y * (1 - xD) + color111.y * xD;
        color11.z = color011.z * (1 - xD) + color111.z * xD;
        color11.w = color011.w * (1 - xD) + color111.w * xD;


        float4 color0, color1;

        color0.x = color00.x * (1 - yD) + color10.x * yD;
        color0.y = color00.y * (1 - yD) + color10.y * yD;
        color0.z = color00.z * (1 - yD) + color10.z * yD;
        color0.w = color00.w * (1 - yD) + color10.w * yD;

        color1.x = color01.x * (1 - yD) + color11.x * yD;
        color1.y = color01.y * (1 - yD) + color11.y * yD;
        color1.z = color01.z * (1 - yD) + color11.z * yD;
        color1.w = color01.w * (1 - yD) + color11.w * yD;

        float4 color;

        color.x = color0.x * (1 - zD) + color1.x * zD;
        color.y = color0.y * (1 - zD) + color1.y * zD;
        color.z = color0.z * (1 - zD) + color1.z * zD;
        color.w = color0.w * (1 - zD) + color1.w * zD;

        return color;
    }
    else{
        return sampleAt(index0);
    }
}
