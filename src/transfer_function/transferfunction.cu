#include "hip/hip_runtime.h"
#include "transferfunction.cuh"

#include <cmath>
#include "quaternion/quaternioncalculator.cuh"


using namespace quaternion;
using namespace grid;
using namespace transfer_function;






__host__ __device__ TransferFunction::TransferFunction(unsigned x, unsigned y, unsigned z) : Grid3D(x, y, z) {
    m_colors = nullptr;
    m_normals = nullptr;

    m_bgColor.x = 1.0f;
    m_bgColor.y = 1.0f;
    m_bgColor.z = 1.0f;
    m_bgColor.w = 1.0f;
}





__host__ __device__ TransferFunction::TransferFunction(const uint3& dim) : Grid3D(dim) {
    m_colors = nullptr;
    m_normals = nullptr;

    m_bgColor.x = 1.0f;
    m_bgColor.y = 1.0f;
    m_bgColor.z = 1.0f;
    m_bgColor.w = 1.0f;
}




__host__ __device__ TransferFunction::~TransferFunction() {
#ifndef __CUDA_ARCH__
    delete []  m_colors;
    m_colors = nullptr;

    delete [] m_normals;
    m_normals = nullptr;
#endif
}


__host__ __device__ float4 TransferFunction::colorAt(float x, float y, float z, const float3& lightPos, const Illumination &illumination, const Interpolation &filterType) const {
    float3 pos;
    pos.x = x;
    pos.y = y;
    pos.z = z;

    return colorAt(pos, lightPos, illumination, filterType);
}




__host__ __device__ float4 TransferFunction::colorAt(const float3& pos, const float3& lightPos, const Illumination &illumSpecs, const Interpolation &interpSpecs) const {
    float4 color;

    if(interpSpecs == Interpolation::TRILINEAR) {
        color = trilinearColorAt(pos);
    }
    else {
        color = nearestColorAt(pos);
    }

    if (illumSpecs.enabled){
        float3 illum = blinnPhongIllum(pos, lightPos, illumSpecs);

        color.x = fminf(1.0f, color.x * illum.x + color.x * illum.y + illum.z);
        color.y = fminf(1.0f, color.y * illum.x + color.y * illum.y + illum.z);
        color.z = fminf(1.0f, color.z * illum.x + color.z * illum.y + illum.z);
    }


    return color;
}





__host__ __device__ float4 TransferFunction::sampleAt(int x, int y, int z) const {
    return sampleAt(index3toIndex(x,y,z));
}




__host__ __device__ float4 TransferFunction::sampleAt(const int3& index) const {
    return sampleAt(index3toIndex(index));
}



__host__ __device__ float4 TransferFunction::sampleAt(size_t index) const {
    uchar4 color = *(m_colors + index);
    float4 colorF;


    colorF.x = color.x / 255.0f;
    colorF.y = color.y / 255.0f;
    colorF.z = color.z / 255.0f;
    colorF.w = color.w / 255.0f;

    return colorF;
}



__host__ __device__ float3 TransferFunction::blinnPhongIllum(const float3& pos, const float3& lightPos, const Illumination &illumination) const {
    int3 index3 = roundPosToIndex(pos);

    float3 normal = *(m_normals + index3toIndex(index3));

    float3 lightDir;
    lightDir.x = (lightPos.x - pos.x);
    lightDir.y = (lightPos.y - pos.y);
    lightDir.z = (lightPos.z - pos.z);
    lightDir = QuaternionCalculator::normalize(lightDir);

    float  diffuseLight = fmaxf(QuaternionCalculator::dotProduct(normal, lightDir), 0.0f);

    float ambient = illumination.ambientPower;
    float diffuse = illumination.diffusePower * diffuseLight;
    float specular = 0.0f;

    if(diffuseLight > 0){
        specular = illumination.specularPower * powf(diffuseLight, illumination.shininess);
    }


    float3 light;

    light.x = ambient;
    light.y = diffuse;
    light.z = specular;

    return light;

}



__host__ __device__ void TransferFunction::setSampleAt(int x, int y, int z, const uchar4 &value) {
    *(m_colors + index3toIndex(x, y , z)) = value;
}






__host__ __device__ void TransferFunction::setSampleAt(const int3& index, const uchar4 &value) {
    *(m_colors + index3toIndex(index)) = value;
}




__host__ __device__ void TransferFunction::setSampleAt(size_t index, const uchar4 &value) {
    *(m_colors + index) = value;

}






__host__ __device__ const float4& TransferFunction::backgroundColor() const {
    return m_bgColor;
}





__host__ __device__ void TransferFunction::setBackGroundColor(const float4& bgColor) {
    m_bgColor = bgColor;
}






__host__ __device__ void TransferFunction::setBackGroundColor(const uchar4 &bgColor) {
    m_bgColor.x = bgColor.x / 255.0f;
    m_bgColor.y = bgColor.y / 255.0f;
    m_bgColor.z = bgColor.z / 255.0f;
    m_bgColor.w = bgColor.w / 255.0f;
}






__host__ __device__ void TransferFunction::setBackGroundColor(float red, float green, float blue, float opacity) {
    m_bgColor.x = red;
    m_bgColor.y = green;
    m_bgColor.z = blue;
    m_bgColor.w = opacity;
}






__host__ __device__ void TransferFunction::setBackGroundColor(std::byte red, std::byte green, std::byte blue, std::byte opacity) {
    m_bgColor.x = static_cast<unsigned char>(red) / 255.0f;
    m_bgColor.y = static_cast<unsigned char>(green) / 255.0f;
    m_bgColor.z = static_cast<unsigned char>(blue) / 255.0f;
    m_bgColor.w = static_cast<unsigned char>(opacity)/ 255.0f;
}





__host__ __device__ float4 TransferFunction::nearestColorAt(const float3& pos) const {
    return sampleAt(roundPosToIndex(pos));
}





__host__ __device__ float4 TransferFunction::trilinearColorAt(const float3& pos) const {
    int3 index = floorPosToIndex(pos);

    if(isIndex3In(index.x, index.y, index.z) && isIndex3In(index.x + 1, index.y + 1, index.z + 1)){
        float4 color = {0, 0, 0, 0};

        float x, y, z;

        x = pos.x - index.x * spacing().x;
        y = pos.y - index.y * spacing().y;
        z = pos.z - index.z * spacing().z;

        float4 color000, color001, color010, color011, color100, color101, color110, color111;

        color000 = sampleAt(index.x, index.y, index.z);
        color001 = sampleAt(index.x, index.y, index.z + 1);
        color010 = sampleAt(index.x, index.y + 1, index.z);
        color011 = sampleAt(index.x, index.y + 1, index.z + 1);
        color100 = sampleAt(index.x + 1, index.y, index.z);
        color101 = sampleAt(index.x + 1, index.y, index.z + 1);
        color110 = sampleAt(index.x + 1, index.y + 1, index.z);
        color111 = sampleAt(index.x + 1, index.y + 1, index.z + 1);



        color.x =  (1 - x) * (1 - y) * (1 - z) * color000.x +
                        (1 - x) * (1 - y) * (z) * color001.x +
                        (1 - x) * (y) * (1 - z) * color010.x +
                        (1 - x) * (y) * (z) * color011.x +
                        (x) * (1 - y) * (1 - z) * color100.x +
                        (x) * (1 - y) * (z) * color101.x +
                        (x) * (y) * (1 - z) * color110.x +
                        (x) * (y) * (z) * color111.x;

        color.y =  (1 - x) * (1 - y) * (1 - z) * color000.y +
                        (1 - x) * (1 - y) * (z) * color001.y +
                        (1 - x) * (y) * (1 - z) * color010.y +
                        (1 - x) * (y) * (z) * color011.y +
                        (x) * (1 - y) * (1 - z) * color100.y +
                        (x) * (1 - y) * (z) * color101.y +
                        (x) * (y) * (1 - z) * color110.y +
                        (x) * (y) * (z) * color111.y;

        color.z = (1 - x) * (1 - y) * (1 - z) * color000.z +
                       (1 - x) * (1 - y) * (z) * color001.z +
                       (1 - x) * (y) * (1 - z) * color010.z +
                       (1 - x) * (y) * (z) * color011.z +
                       (x) * (1 - y) * (1 - z) * color100.z +
                       (x) * (1 - y) * (z) * color101.z +
                       (x) * (y) * (1 - z) * color110.z +
                       (x) * (y) * (z) * color111.z;

        color.w = (1 - x) * (1 - y) * (1 - z) * color000.w +
                       (1 - x) * (1 - y) * (z) * color001.w +
                       (1 - x) * (y) * (1 - z) * color010.w +
                       (1 - x) * (y) * (z) * color011.w +
                       (x) * (1 - y) * (1 - z) * color100.w +
                       (x) * (1 - y) * (z) * color101.w +
                       (x) * (y) * (1 - z) * color110.w +
                       (x) * (y) * (z) * color111.w;

        return color;
    }
    else{
        return sampleAt(index);
    }
}
