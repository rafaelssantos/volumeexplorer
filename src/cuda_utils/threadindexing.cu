#include "threadindexing.cuh"


#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <>









__device__ unsigned ThreadIndexing::globalIdx() {
    unsigned blockId;
    unsigned id;

    blockId = blockIdx.x + (blockIdx.y * gridDim.x) + (gridDim.x * gridDim.y * blockIdx.z);
    id = blockId * (blockDim.x * blockDim.y * blockDim.z) + (threadIdx.z * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;

    return id;
}





__device__ unsigned ThreadIndexing::totalBlockIdx() {
    return (threadIdx.z * blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
}





__device__ unsigned ThreadIndexing::totalBlockDim() {
    return blockDim.x * blockDim.y * blockDim.z;
}
