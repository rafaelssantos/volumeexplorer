#include "hip/hip_runtime.h"
#include "raycasting.cuh"


#include <>
#include "cuda_utils/kerneldimscalculator.h"
#include "cuda_utils/hip/hip_runtime_api.h"
#include "cuda_utils/threadindexing.cuh"
#include "quaternion/quaternioncalculator.cuh"




using namespace quaternion;
using namespace rendering::view;
using namespace rendering::image;
using namespace rendering;
using namespace rendering::ray_casting;
using namespace transfer_function;
using namespace volume;





__global__ void _rayCastingKernel(GlPBOImage image, const Volume* volume, const TransferFunction* transferFunc, const RenderingSpecs renderingSpecs, const Viewer viewer, const Illumination illumination, const Interpolation interpolation) {
    const unsigned tid = ThreadIndexing::globalIdx();


    if(tid < image.pixelCount()) {
        uint2 pixel;        //Pixel associado à thread atual
        pixel.x = tid % image.size().x;
        pixel.y = tid / image.size().x;

        float3 eye;     /* Posição do observador associada à thread atual. Projeção ortogonal em relação ao plano de imagem.Será iterada até entrar e sair do volume.*/
        eye.x = pixel.x * image.pixelAspect() - (image.pixelAspect() * image.size().x - volume->size().x) / 2.0f;
        eye.y = pixel.y * image.pixelAspect() - (image.pixelAspect() * image.size().y - volume->size().y) / 2.0f;
        eye.z = volume->center().z + volume->diameter() / 2.0f + renderingSpecs.step;

        float3 pos;
        pos.x = eye.x;
        pos.y = eye.y;
        pos.z = eye.z - renderingSpecs.step;

        eye = QuaternionCalculator::rotate(eye, viewer.rotation, volume->center());
        pos = QuaternionCalculator::rotate(pos, viewer.rotation, volume->center());

        unsigned nSteps = 1;
        float3 step;
        step.x = pos.x - eye.x;
        step.y = pos.y - eye.y;
        step.z = pos.z - eye.z;

        float4 colorOut;  //Cor acumulada que será atribuida ao pixel
        colorOut.x = 0;   //Vermelho
        colorOut.y = 0;   //verde
        colorOut.z = 0;   //Azul
        colorOut.w = 0;   //Opacidade

        float opacityCorrFactor = renderingSpecs.step / volume->voxelDiameter();

        while(nSteps <= renderingSpecs.maxSteps && volume->isInCircumsphere(pos)) {                                                  //Dentro da esfera circunscrita ao volume
            if(volume->isPosIn(pos)) {
                while(nSteps <= renderingSpecs.maxSteps && volume->isPosIn(pos) && colorOut.w <= renderingSpecs.maxOpacity) {       //Dentro do volume
                    float4 colorIn = transferFunc->colorAt(pos, eye, illumination, interpolation);

                    colorIn.w = 1.0f - powf(1.0f - colorIn.w, opacityCorrFactor);

                    colorOut.x = colorOut.x + colorIn.x * colorIn.w * (1 - colorOut.w);
                    colorOut.y = colorOut.y + colorIn.y * colorIn.w * (1 - colorOut.w);
                    colorOut.z = colorOut.z + colorIn.z * colorIn.w * (1 - colorOut.w);
                    colorOut.w = colorOut.w + colorIn.w * (1 - colorOut.w);

                    nSteps++;
                    pos.x = ((nSteps * renderingSpecs.step) * step.x) / renderingSpecs.step + eye.x;
                    pos.y = ((nSteps * renderingSpecs.step) * step.y) / renderingSpecs.step + eye.y;
                    pos.z = ((nSteps * renderingSpecs.step) * step.z) / renderingSpecs.step + eye.z;
                }
                break;
            }

            nSteps++;
            pos.x = ((nSteps * renderingSpecs.step) * step.x) / renderingSpecs.step + eye.x;
            pos.y = ((nSteps * renderingSpecs.step) * step.y) / renderingSpecs.step + eye.y;
            pos.z = ((nSteps * renderingSpecs.step) * step.z) / renderingSpecs.step + eye.z;
        }

        colorOut.x += (transferFunc->backgroundColor().x * transferFunc->backgroundColor().w * (1.0f - colorOut.w));
        colorOut.y += (transferFunc->backgroundColor().y * transferFunc->backgroundColor().w * (1.0f - colorOut.w));
        colorOut.z += (transferFunc->backgroundColor().z * transferFunc->backgroundColor().w * (1.0f - colorOut.w));
        colorOut.w += transferFunc->backgroundColor().w  * (1.0f - colorOut.w);

        uchar4 accumByteColor;

        accumByteColor.x = static_cast<unsigned char>(colorOut.x * 255.0f);
        accumByteColor.y = static_cast<unsigned char>(colorOut.y * 255.0f);
        accumByteColor.z = static_cast<unsigned char>(colorOut.z * 255.0f);
        accumByteColor.w = static_cast<unsigned char>(colorOut.w * 255.0f);

        image.setPixelColor(pixel.x, pixel.y, accumByteColor);
    }
}





RayCasting::RayCasting() {

}





RayCasting::~RayCasting() {

}





RayCasting& RayCasting::instance() {
    static RayCasting instance;

    return instance;
}





void RayCasting::kernelLaunch(GlPBOImage& image, const Volume* volume, const TransferFunction* transferFunc, const RenderingSpecs& renderingSpecs, const view::Viewer& observer, dim3 blockDim, unsigned threadsCount, const Illumination& illumination, const Interpolation& interpolation) {
    dim3 gridDim;
    gridDim = KernelDimsCalculator::instance().calc(blockDim, threadsCount);

    _rayCastingKernel<<<gridDim, blockDim>>>(image, volume, transferFunc, renderingSpecs, observer, illumination, interpolation);
    getLastCudaError("Ray casting kernel failed.");
}
