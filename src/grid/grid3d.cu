#include "hip/hip_runtime.h"
#include "grid3d.cuh"


#include <cmath>



using namespace std;
using namespace grid;





__host__ __device__ Grid3D::Grid3D(unsigned x, unsigned y, unsigned z) {
    m_dim.x = x;
    m_dim.y = y;
    m_dim.z = z;
	m_spacing.x = m_spacing.y = m_spacing.z = 1;
    m_size.x = m_size.y = m_size.z = 0;
    m_center.x = m_center.y = m_center.z = 0;
    m_diameter = 0;
    m_voxelDiameter = 0;
}





__host__ __device__ Grid3D::Grid3D(const uint3& dim) {
    m_dim = dim;
	m_spacing.x = m_spacing.y = m_spacing.z = 1;
    m_size.x = m_size.y = m_size.z = 0;
    m_center.x = m_center.y = m_center.z = 0;
    m_diameter = 0;
    m_voxelDiameter = 0;
}





__host__ __device__ Grid3D::~Grid3D() {

}





__host__ __device__ void Grid3D::setSpacing(const float3& spacing) {
	m_spacing = spacing;

    measure();
}





__host__ __device__ void Grid3D::setSpacing(float x, float y, float z) {
	m_spacing.x = x;
	m_spacing.y = y;
	m_spacing.z = z;

    measure();
}





__host__ __device__ const float3& Grid3D::spacing() const {
	return m_spacing;
}





__host__ __device__ const uint3& Grid3D::dim() const {
    return m_dim;
}





__host__ __device__ unsigned int Grid3D::voxelCount() const {
    return m_dim.x * m_dim.y * m_dim.z;
}





__host__ __device__ const float3& Grid3D::size() const {
    return m_size;
}





__host__ __device__ int Grid3D::index3toIndex(int x, int y, int z) const {
    if(x < 0 || x >= m_dim.x || y < 0 || y >= m_dim.y || z < 0 || z >= m_dim.z){
        return -1;
    }
    else{
        return z * static_cast<int>(m_dim.y * m_dim.x) + y * static_cast<int>(m_dim.x) + x;
    }
}





__host__ __device__ int Grid3D::index3toIndex(const int3& index) const {
    if(index.x < 0 || index.x >= m_dim.x || index.y < 0 || index.y >= m_dim.y || index.z < 0 || index.z >= m_dim.z){
        return -1;
    }
    else{
        return index.z * static_cast<int>(m_dim.y * m_dim.x) + index.y * static_cast<int>(m_dim.x) + index.x;
    }
}





__host__ __device__ int3 Grid3D::indexToIndex3(int index) const {
    int3 int3Index;

    int3Index.z = (index / static_cast<int>(m_dim.y * m_dim.x));
    int3Index.y = (index % static_cast<int>(m_dim.y * m_dim.x)) / static_cast<int>(m_dim.x);
    int3Index.x = (index % static_cast<int>(m_dim.y * m_dim.x)) % static_cast<int>(m_dim.x);

    return int3Index;
}





__host__ __device__ int3 Grid3D::floorPosToIndex(const float3& pos) const {
    int3 index;

	index.x = static_cast<int>(pos.x /spacing().x);
	index.y = static_cast<int>(pos.y /spacing().y);
	index.z = static_cast<int>(pos.z /spacing().z);

    return index;
}




__host__ __device__ int3 Grid3D::floorPosToIndex(float x, float y, float z) const {
    int3 index;

	index.x = static_cast<int>(x /spacing().x);
	index.y = static_cast<int>(y /spacing().y);
	index.z = static_cast<int>(z /spacing().z);

    return index;
}





__host__ __device__ int3 Grid3D::roundPosToIndex(const float3& pos) const {
    int3 index;

	index.x = static_cast<int>(roundf(pos.x / spacing().x));
	index.y = static_cast<int>(roundf(pos.y / spacing().y));
	index.z = static_cast<int>(roundf(pos.z / spacing().z));

    return index;
}




__host__ __device__ int3 Grid3D::roundPosToIndex(float x, float y, float z) const {
    int3 index;

	index.x = static_cast<int>(roundf(x / spacing().x));
	index.y = static_cast<int>(roundf(y / spacing().y));
	index.z = static_cast<int>(roundf(z / spacing().z));

    return index;
}




__host__ __device__ int3 Grid3D::ceilPosToIndex(const float3& pos) const {
    int3 index;

	index.x = static_cast<int>(ceilf(pos.x / spacing().x));
	index.y = static_cast<int>(ceilf(pos.y / spacing().y));
	index.z = static_cast<int>(ceilf(pos.z / spacing().z));

    return index;
}




__host__ __device__ int3 Grid3D::ceilPosToIndex(float x, float y, float z) const {
    int3 index;

	index.x = static_cast<int>(ceilf(x / spacing().x));
	index.y = static_cast<int>(ceilf(y / spacing().y));
	index.z = static_cast<int>(ceilf(z / spacing().z));

    return index;
}





__host__ __device__ bool Grid3D::isPosIn(float x, float y, float z) const {
    if(x < 0 || y < 0 || z < 0 || x > m_size.x || y > m_size.y || z > m_size.z) {
        return false;
    }
    else{
        return true;
    }
}





__host__ __device__ bool Grid3D::isPosIn(const float3& pos) const {
    if(pos.x < 0 || pos.y < 0 || pos.z < 0 || pos.x > m_size.x || pos.y > m_size.y || pos.z > m_size.z) {
        return false;
    }
    else{
        return true;
    }
}






__host__ __device__ bool Grid3D::isIndex3In(int x, int y, int z) const {
    if(x < 0 || y < 0 || z < 0 || x >= m_dim.x || y >= m_dim.y|| z >= m_dim.z) {
        return false;
    }
    else{
        return true;
    }
}





__host__ __device__ bool Grid3D::isIndex3In(const int3& index) const {
    if(index.x < 0 || index.y < 0 || index.z < 0 || index.x >= m_dim.x || index.y >= m_dim.y || index.z >= m_dim.z) {
        return false;
    }
    else{
        return true;
    }
}



__host__ __device__ float Grid3D::diameter() const {
    return m_diameter;
}




__host__ __device__ float Grid3D::voxelDiameter() const {
    return m_voxelDiameter;
}



__host__ __device__ const float3 &Grid3D::center() const {
    return m_center;
}





string Grid3D::toString() {
    string description;

	description = "Grid: " + to_string(m_dim.x) + "×" + to_string(m_dim.y) + "×" + to_string(m_dim.z) + " | " ;
    description += "Spacing: " + to_string(m_spacing.x) + "×" + to_string(m_spacing.y) + "×" + to_string(m_spacing.z);

    return description;
}





__host__ __device__ void Grid3D::measure() {
	m_size.x = (m_dim.x - 1) * m_spacing.x;
	m_size.y = (m_dim.y - 1) * m_spacing.y;
	m_size.z = (m_dim.z - 1) * m_spacing.z;

    m_center = {m_size.x / 2.0f, m_size.y / 2.0f, m_size.z / 2.0f};
    m_diameter = sqrtf(m_size.x * m_size.x + m_size.y * m_size.y + m_size.z * m_size.z);
    m_voxelDiameter = sqrtf(m_spacing.x * m_spacing.x + m_spacing.y * m_spacing.y + m_spacing.z * m_spacing.z);
}
