#include "hip/hip_runtime.h"
#include "tfmanager.cuh"

#include <hip/hip_runtime_api.h>
#include <>
#include "cuda_utils/kerneldimscalculator.h"
#include "quaternion/quaternioncalculator.cuh"
#include "cuda_utils/threadindexing.cuh"
#include "cuda_utils/hip/hip_runtime_api.h"
#include "settings/renderingsettings.h"
#include "tfmemmanager.h"




using namespace quaternion;
using namespace transfer_function;
using namespace volume;
using namespace settings;
using namespace std;



__global__ void _calcNormalsKernel(const Volume* volume, TransferFunction *transferFunc) {
    const unsigned tid = ThreadIndexing::globalIdx();

    if(tid < volume->voxelCount()) {
        TransferFunctionManager::_sobelOperator(volume, transferFunc, static_cast<int>(tid));
    }
}





TransferFunctionManager& TransferFunctionManager::instance() {
    static TransferFunctionManager instance;

    return instance;
}





std::pair<TransferFunction*, TransferFunction*> TransferFunctionManager::build(const Volume* hostVolume, const Volume* devVolume) const {
    TransferFunction* hostTransferFunc = new TransferFunction(hostVolume->dim());
    hostTransferFunc->setSpacing(hostVolume->spacing());

    hostTransferFunc->m_colors = new uchar4[hostVolume->dim().x * hostVolume->dim().y * hostVolume->dim().z];
    hostTransferFunc->m_normals = new float3[hostVolume->dim().x * hostVolume->dim().y * hostVolume->dim().z];

    for(auto x = 0; static_cast<unsigned>(x) < hostTransferFunc->dim().x; x++) {
        for(auto y = 0; static_cast<unsigned>(y) < hostTransferFunc->dim().y; y++) {
            for(auto z = 0; static_cast<unsigned>(z) < hostTransferFunc->dim().z; z++) {
                uchar4 color;
                color.x = color.y = color.z = color.w = static_cast<unsigned char>(hostVolume->sampleAt(x, y, z));
                hostTransferFunc->setSampleAt(x, y, z, color);
            }
        }
    }

    TransferFunction* devTransferFunc = TFMemManager::instance().cloneToDev(hostTransferFunc);

    dim3 blockDim = RenderingSettings::instance().blockDim();
    dim3 gridDim = KernelDimsCalculator::instance().calc(blockDim, hostVolume->voxelCount());

    _calcNormalsKernel<<<gridDim, RenderingSettings::instance().blockDim()>>>(devVolume, devTransferFunc);

    getLastCudaError("Transfer function normals kernel failed.");
    TFMemManager::instance().memcpyNormalsFromBuffer(hostTransferFunc);


    return std::pair<TransferFunction*, TransferFunction*>(hostTransferFunc, devTransferFunc);
}





void TransferFunctionManager::free(std::pair<TransferFunction*, TransferFunction*> transferFunc) const {
    TFMemManager::instance().freeHost(transferFunc.first);
    TFMemManager::instance().freeDevice(transferFunc.second);
}





TransferFunctionManager::TransferFunctionManager() {

}




TransferFunctionManager::~TransferFunctionManager() {

}




__host__ __device__ void TransferFunctionManager::_sobelOperator(const Volume* volume, TransferFunction* transferFunc, int index) {
    int3 voxelIndex = volume->indexToIndex3(index);

    float gX[3][3][3], gY[3][3][3], gZ[3][3][3];
    float hX[3] = {+1, +2, +1}, hY[3] = {+1, +2, +1}, hZ[3] = {+1, +2, +1};
    float hpX[3] = {+1, +0, -1}, hpY[3] = {+1, +0, -1}, hpZ[3] = {+1, +0,-1};

    for(int m = 0; m <= 2; m++){
        for(int n = 0; n <= 2; n++){
            for(int k = 0; k <= 2; k++){
                gY[m][n][k] = hpX[m] * hY[n] * hZ[k];
                gX[m][n][k] = hX[m] * hpY[n] * hZ[k];
                gZ[m][n][k] = hX[m] * hY[n] * hpZ[k];
            }
        }
    }


    float3 gradient = {0, 0, 0};

    for(int m = -1; m <= 1; m++){
        for(int n = -1; n <= 1; n++){
            for(int k = -1; k <= 1; k++){
                if(volume->isIndex3In(voxelIndex.x - m, voxelIndex.y - n, voxelIndex.z - k)){
                    gradient.x += gY[m+1][n+1][k+1] * static_cast<float>(volume->sampleAt(voxelIndex.x - m, voxelIndex.y - n, voxelIndex.z - k));
                    gradient.y += gX[m+1][n+1][k+1] * static_cast<float>(volume->sampleAt(voxelIndex.x - m, voxelIndex.y - n, voxelIndex.z - k));
                    gradient.z += gZ[m+1][n+1][k+1] * static_cast<float>(volume->sampleAt(voxelIndex.x - m, voxelIndex.y - n, voxelIndex.z - k));
                }
            }
        }
    }

    gradient.x = -gradient.x;
    gradient.y = -gradient.y;
    gradient.z = -gradient.z;

    *(transferFunc->m_normals + index) = QuaternionCalculator::normalize(gradient);
}




__host__ __device__ void TransferFunctionManager::_centralDifferences(const volume::Volume *volume, TransferFunction *transferFunc, int index) {
    int3 index3 = volume->indexToIndex3(index);
    float3 gradient = {0, 0, 0};

    if(index3.x - 1 >= 0 && index3.x + 1 <transferFunc->dim().x){
        gradient.x = (static_cast<float>(volume->sampleAt(index3.x + 1, index3.y, index3.z)) - static_cast<float>(volume->sampleAt(index3.x - 1, index3.y, index3.z))) / (2 * volume->spacing().x);
    }
    if(index3.y - 1 >= 0 && index3.y + 1 <transferFunc->dim().y){
        gradient.y = (static_cast<float>(volume->sampleAt(index3.x, index3.y + 1, index3.z)) - static_cast<float>(volume->sampleAt(index3.x, index3.y - 1, index3.z))) / (2 * volume->spacing().y);
    }
    if(index3.z - 1 >= 0 && index3.z + 1 < transferFunc->dim().z){
        gradient.z = (static_cast<float>(volume->sampleAt(index3.x, index3.y, index3.z + 1)) - static_cast<float>(volume->sampleAt(index3.x, index3.y, index3.z - 1))) / (2  *volume->spacing().z);
    }

    *(transferFunc->m_normals + index) = QuaternionCalculator::normalize(gradient);
}





void TransferFunctionManager::memcpyColors(TransferFunction* hostTransferFunc, TransferFunction* devTransferFunc) {
    TFMemManager::instance().memcpyColors(devTransferFunc, hostTransferFunc);
}
